#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2016
 *
 * Cristián Maureira-Fredes <cmaureirafredes@gmail.com>
 *
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *
 * 1. Redistributions of source code must retain the above copyright
 * notice, this list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright
 * notice, this list of conditions and the following disclaimer in the
 * documentation and/or other materials provided with the distribution.
 *
 * 3. The name of the author may not be used to endorse or promote
 * products derived from this software without specific prior written
 * permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE AUTHOR ``AS IS'' AND ANY EXPRESS
 * OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED.  IN NO EVENT SHALL THE AUTHOR BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE
 * GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER
 * IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR
 * OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN
 * IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 */
#undef _GLIBCXX_ATOMIC_BUILTINS
#include "Hermite4GPU.cuh"

/** Constructor that uses its parent one.
 * Additionally handles the split of the particles of the system among the available
 * GPUs, allocation of the variables, and defining widely use sizes for arrays.
 */
Hermite4GPU::Hermite4GPU(NbodySystem *ns, Logger *logger, NbodyUtils *nu)
            : Hermite4(ns, logger, nu)
{
    smem = sizeof(Predictor) * BSIZE;
    smem_reduce = sizeof(Forces) * NJBLOCK + 1;

    int detected_gpus;
    CSC(hipGetDeviceCount(&detected_gpus));

    if (ns->gpus > 0)
    {
        gpus = ns->gpus;
    }
    else
    {
        gpus = detected_gpus;
    }

    if (detected_gpus > gpus)
    {
        std::string s = "";
        s += std::string("Not using all the available GPUs: ");
        s += std::string(SSTR(gpus));
        s += std::string(" of ");
        s += std::string(SSTR(detected_gpus));
        logger->log_warning(s);
    }

    logger->log_info(std::string("GPUs: ")+std::string(SSTR(gpus)));

    std::string ss = "";
    ss += std::string("Splitting ");
    ss += std::string(SSTR(ns->n));
    ss += std::string(" particles in ");
    ss += std::string(SSTR(gpus));
    ss += std::string(" GPUs");
    logger->log_info(ss);

    if (ns->n % gpus == 0)
    {
        size_t size = ns->n/gpus;
        for ( int g = 0; g < gpus; g++)
            n_part[g] = size;
    }
    else
    {
        size_t size = std::ceil(ns->n/(float)gpus);
        for ( int g = 0; g < gpus; g++)
        {
            if (ns->n - size*(g+1) > 0)
                n_part[g] = size;
            else
                n_part[g] = ns->n - size*g;
        }
    }

    for(int g = 0; g < gpus; g++)
    {
        std::string sss = "";
        sss += std::string("GPU ");
        sss += std::string(SSTR(g));
        sss += std::string(" particles: ");
        sss += std::string(SSTR(n_part[g]));
        logger->log_info(sss);
    }

    i1_size = ns->n * sizeof(int);
    d1_size = ns->n * sizeof(double);
    d4_size = ns->n * sizeof(double4);
    ff_size = ns->n * sizeof(Forces);
    pp_size = ns->n * sizeof(Predictor);

    alloc_arrays_device();
}

/** Destructor in charge of memory deallocation */
Hermite4GPU::~Hermite4GPU()
{
    free_arrays_device();
}

/** Method in charge of allocating the data structures on the available GPUs,
 * also initializing all the arrays to zero
 */
void Hermite4GPU::alloc_arrays_device()
{
    for(int g = 0; g < gpus; g++)
    {
        // Setting GPU
        CSC(hipSetDevice(g));

        CSC(hipMalloc((void**)&ns->d_r[g], d4_size));
        CSC(hipMalloc((void**)&ns->d_v[g], d4_size));
        CSC(hipMalloc((void**)&ns->d_f[g], ff_size));
        CSC(hipMalloc((void**)&ns->d_p[g], pp_size));
        CSC(hipMalloc((void**)&ns->d_t[g], d1_size));
        CSC(hipMalloc((void**)&ns->d_i[g], pp_size));
        CSC(hipMalloc((void**)&ns->d_dt[g], d1_size));
        CSC(hipMalloc((void**)&ns->d_ekin[g], d1_size));
        CSC(hipMalloc((void**)&ns->d_epot[g], d1_size));
        CSC(hipMalloc((void**)&ns->d_move[g], i1_size));
        CSC(hipMalloc((void**)&ns->d_fout[g], ff_size * NJBLOCK));
        CSC(hipMalloc((void**)&ns->d_fout_tmp[g], ff_size * NJBLOCK));

        CSC(hipMemset(ns->d_r[g], 0, d4_size));
        CSC(hipMemset(ns->d_v[g], 0, d4_size));
        CSC(hipMemset(ns->d_f[g], 0, ff_size));
        CSC(hipMemset(ns->d_p[g], 0, pp_size));
        CSC(hipMemset(ns->d_t[g], 0, d1_size));
        CSC(hipMemset(ns->d_i[g], 0, pp_size));
        CSC(hipMemset(ns->d_dt[g], 0, d1_size));
        CSC(hipMemset(ns->d_ekin[g], 0, d1_size));
        CSC(hipMemset(ns->d_epot[g], 0, d1_size));
        CSC(hipMemset(ns->d_move[g], 0, i1_size));
        CSC(hipMemset(ns->d_fout[g], 0, ff_size * NJBLOCK));
        CSC(hipMemset(ns->d_fout_tmp[g], 0, ff_size * NJBLOCK));

        ns->h_fout_gpu[g] = new Forces[ns->n*NJBLOCK];
    }

    // Extra CPU array
    ns->h_fout_tmp = new Forces[ns->n*NJBLOCK];
}

/** Method in charge of deallocating the data structures on the available GPUs.
 */
void Hermite4GPU::free_arrays_device()
{

    for(int g = 0; g < gpus; g++)
    {
        // Setting GPU
        CSC(hipSetDevice(g));

        CSC(hipFree(ns->d_r[g]));
        CSC(hipFree(ns->d_v[g]));
        CSC(hipFree(ns->d_f[g]));
        CSC(hipFree(ns->d_p[g]));
        CSC(hipFree(ns->d_t[g]));
        CSC(hipFree(ns->d_i[g]));
        CSC(hipFree(ns->d_dt[g]));
        CSC(hipFree(ns->d_ekin[g]));
        CSC(hipFree(ns->d_epot[g]));
        CSC(hipFree(ns->d_move[g]));
        CSC(hipFree(ns->d_fout[g]));
        CSC(hipFree(ns->d_fout_tmp[g]));
        delete ns->h_fout_gpu[g];
    }

    delete ns->h_fout_tmp;
    //delete ns->h_fout_gpu;
}

/** Method in charge of the prediction step.
 * This can be use on the CPU (commented section) or on the GPUs.
 * The reason of having both reasons, is the improvement is not much for small
 * amount of particles.
 */
void Hermite4GPU::predicted_pos_vel(double ITIME)
{
    ns->gtime.prediction_ini = omp_get_wtime();
    //#pragma omp parallel for
    //for (int i = 0; i < ns->n; i++)
    //{
    //    double dt  = ITIME - ns->h_t[i];
    //    double dt2 = 0.5*(dt  * dt);
    //    double dt3 = 0.166666666666666*(dt * dt * dt);

    //    Forces ff = ns->h_f[i];
    //    double4 rr = ns->h_r[i];
    //    double4 vv = ns->h_v[i];

    //    ns->h_p[i].r[0] = (dt3 * ff.a1[0]) + (dt2 * ff.a[0]) + (dt * vv.x) + rr.x;
    //    ns->h_p[i].r[1] = (dt3 * ff.a1[1]) + (dt2 * ff.a[1]) + (dt * vv.y) + rr.y;
    //    ns->h_p[i].r[2] = (dt3 * ff.a1[2]) + (dt2 * ff.a[2]) + (dt * vv.z) + rr.z;

    //    ns->h_p[i].v[0] = (dt2 * ff.a1[0]) + (dt * ff.a[0]) + vv.x;
    //    ns->h_p[i].v[1] = (dt2 * ff.a1[1]) + (dt * ff.a[1]) + vv.y;
    //    ns->h_p[i].v[2] = (dt2 * ff.a1[2]) + (dt * ff.a[2]) + vv.z;

    //    ns->h_p[i].m = rr.w;
    //}

    for(int g = 0; g < gpus; g++)
    {
        CSC(hipSetDevice(g));
        int shift = g*n_part[g-1];
        size_t ff_size = n_part[g] * sizeof(Forces);
        size_t d4_size = n_part[g] * sizeof(double4);
        size_t d1_size = n_part[g] * sizeof(double);

        CSC(hipMemcpyAsync(ns->d_f[g], ns->h_f + shift, ff_size, hipMemcpyHostToDevice, 0));
        CSC(hipMemcpyAsync(ns->d_r[g], ns->h_r + shift, d4_size, hipMemcpyHostToDevice, 0));
        CSC(hipMemcpyAsync(ns->d_v[g], ns->h_v + shift, d4_size, hipMemcpyHostToDevice, 0));
        CSC(hipMemcpyAsync(ns->d_t[g], ns->h_t + shift, d1_size, hipMemcpyHostToDevice, 0));
    }

    // Executing kernels
    for(int g = 0; g < gpus; g++)
    {
        CSC(hipSetDevice(g));

        nthreads = BSIZE;
        nblocks = std::ceil(n_part[g]/(float)nthreads);

        k_prediction <<< nblocks, nthreads >>> (ns->d_f[g],
                                                ns->d_r[g],
                                                ns->d_v[g],
                                                ns->d_t[g],
                                                ns->d_p[g],
                                                n_part[g],
                                                ITIME);
        get_kernel_error();
    }

    for(int g = 0; g < gpus; g++)
    {
        CSC(hipSetDevice(g));
        size_t slice = g*n_part[g-1];
        size_t pp_size = n_part[g] * sizeof(Predictor);

        CSC(hipMemcpyAsync(&ns->h_p[slice], ns->d_p[g], pp_size, hipMemcpyDeviceToHost, 0));
    }

    ns->gtime.prediction_end += omp_get_wtime() - ns->gtime.prediction_ini;
}

/** Method in charge of the corrector step.
 * This is not implemented on the GPU because the benefit was not much
 * for small amount of particles.
 */
void Hermite4GPU::correction_pos_vel(double ITIME, int nact)
{
    ns->gtime.correction_ini = omp_get_wtime();
    int i;
    #pragma omp parallel for private(i)
    for (int k = 0; k < nact; k++)
    {
        i = ns->h_move[k];

        Forces ff = ns->h_f[i];
        Forces oo = ns->h_old[i];
        Predictor pp = ns->h_p[i];

        double dt1 = ns->h_dt[i];
        double dt2 = dt1 * dt1;
        double dt3 = dt2 * dt1;
        double dt4 = dt2 * dt2;
        double dt5 = dt4 * dt1;

        double dt2inv = 1.0/dt2;
        double dt3inv = 1.0/dt3;

        double dt3_6 = 0.166666666666666*dt3;
        double dt4_24 = 0.041666666666666*dt4;
        double dt5_120 = 0.008333333333333*dt5;

        // Acceleration 2nd derivate
        ns->h_a2[i].x = (-6 * (oo.a[0] - ff.a[0] ) - dt1 * (4 * oo.a1[0] + 2 * ff.a1[0]) ) * dt2inv;
        ns->h_a2[i].y = (-6 * (oo.a[1] - ff.a[1] ) - dt1 * (4 * oo.a1[1] + 2 * ff.a1[1]) ) * dt2inv;
        ns->h_a2[i].z = (-6 * (oo.a[2] - ff.a[2] ) - dt1 * (4 * oo.a1[2] + 2 * ff.a1[2]) ) * dt2inv;

        // Acceleration 3rd derivate
        ns->h_a3[i].x = (12 * (oo.a[0] - ff.a[0] ) + 6 * dt1 * (oo.a1[0] + ff.a1[0]) ) * dt3inv;
        ns->h_a3[i].y = (12 * (oo.a[1] - ff.a[1] ) + 6 * dt1 * (oo.a1[1] + ff.a1[1]) ) * dt3inv;
        ns->h_a3[i].z = (12 * (oo.a[2] - ff.a[2] ) + 6 * dt1 * (oo.a1[2] + ff.a1[2]) ) * dt3inv;


        // Correcting position
        ns->h_r[i].x = pp.r[0] + (dt4_24)*ns->h_a2[i].x + (dt5_120)*ns->h_a3[i].x;
        ns->h_r[i].y = pp.r[1] + (dt4_24)*ns->h_a2[i].y + (dt5_120)*ns->h_a3[i].y;
        ns->h_r[i].z = pp.r[2] + (dt4_24)*ns->h_a2[i].z + (dt5_120)*ns->h_a3[i].z;

        // Correcting velocity
        ns->h_v[i].x = pp.v[0] + (dt3_6)*ns->h_a2[i].x + (dt4_24)*ns->h_a3[i].x;
        ns->h_v[i].y = pp.v[1] + (dt3_6)*ns->h_a2[i].y + (dt4_24)*ns->h_a3[i].y;
        ns->h_v[i].z = pp.v[2] + (dt3_6)*ns->h_a2[i].z + (dt4_24)*ns->h_a3[i].z;

        ns->h_t[i] = ITIME;

        double normal_dt  = nu->get_timestep_normal(i, ns->eta);
        ns->h_dt[i] = nu->normalize_dt(normal_dt, ns->h_dt[i], ns->h_t[i], i);

    }
    ns->gtime.correction_end += omp_get_wtime() - ns->gtime.correction_ini;
}

/** Method in charge of the initialization of all the particle's acceleration
 * and first derivative of the system, at the begining of the simulation.
 */
void Hermite4GPU::init_acc_jrk()
{
    size_t pp_size = ns->n * sizeof(Predictor);

    // Copying arrays to device
    #pragma omp parallel for num_threads(gpus)
    for(int g = 0; g < gpus; g++)
    {
        CSC(hipSetDevice(g));

        // All this information from the predictors is needed by each device
        CSC(hipMemcpy(ns->d_p[g], ns->h_p, pp_size, hipMemcpyHostToDevice));
        //CSC(hipMemcpyAsync(ns->d_p[g], ns->h_p, pp_size, hipMemcpyHostToDevice, 0));
    }

    // Executing kernels
    for(int g = 0; g < gpus; g++)
    {
        CSC(hipSetDevice(g));

        nthreads = BSIZE;
        nblocks = std::ceil(n_part[g]/(float)nthreads);

        k_init_acc_jrk <<< nblocks, nthreads, smem >>> (ns->d_p[g],
                                                        ns->d_f[g],
                                                        ns->n,
                                                        ns->e2,
                                                        g,
                                                        n_part[g]);
        get_kernel_error();
    }

    for(int g = 0; g < gpus; g++)
    {
        CSC(hipSetDevice(g));

        size_t chunk = n_part[g]*sizeof(Forces);
        size_t slice = g*n_part[g-1];

        CSC(hipMemcpy(&ns->h_f[slice], ns->d_f[g], chunk, hipMemcpyDeviceToHost));
        //CSC(hipMemcpyAsync(&ns->h_f[slice], ns->d_f[g], chunk, hipMemcpyDeviceToHost, 0));
    }
}

/** Method in charge of the force interaction between \f$N_{act}\f$ and the whole
 * system.
 *  First there is a tmp construction of predictors to be send to the GPUs.
 *  Then the data is copied to the devices.
 *  The first kernel perform the preliminary calculation of the forces in JPBLOCKS.
 *  The second kernel, reduction, is in charge of summing all the preliminary forces
 *  to the final value for all the active particles.
 */
void Hermite4GPU::update_acc_jrk(int nact)
{
    // Timer begin
    ns->gtime.update_ini = omp_get_wtime();

    //for(int g = 0; g < gpus; g++)
    //{
    //    if (n_part[g] > 0)
    //    {
    //        size_t pp_size = n_part[g] * sizeof(Predictor);
    //        int shift = g*n_part[g-1];

    //        CSC(hipSetDevice(g));
    //        // Copying to the device the predicted r and v
    //        //CSC(hipMemcpy(ns->d_p[g], ns->h_p + shift, pp_size, hipMemcpyHostToDevice));
    //        CSC(hipMemcpyAsync(ns->d_p[g], ns->h_p + shift, pp_size, hipMemcpyHostToDevice, 0));
    //    }
    //}

    // Fill the h_i Predictor array with the particles that we need to move
    #pragma omp parallel for
    for (int i = 0; i < nact; i++)
    {
        ns->h_i[i] = ns->h_p[ns->h_move[i]];
    }

    for(int g = 0; g < gpus; g++)
    {
        if (n_part[g] > 0)
        {
            CSC(hipSetDevice(g));
            // Copy to the GPU (d_i) the preddictor host array (h_i)
            size_t chunk = nact * sizeof(Predictor);
            //CSC(hipMemcpy(ns->d_i[g], ns->h_i, chunk, hipMemcpyHostToDevice));
            CSC(hipMemcpyAsync(ns->d_i[g], ns->h_i, chunk, hipMemcpyHostToDevice, 0));
        }
    }

    ns->gtime.grav_ini = omp_get_wtime();
    for(int g = 0; g < gpus; g++)
    {
        if (n_part[g] > 0)
        {
            CSC(hipSetDevice(g));
            // Blocks, threads and shared memory configuration
            int  nact_blocks = 1 + (nact-1)/BSIZE;
            dim3 nblocks(nact_blocks, NJBLOCK, 1);
            dim3 nthreads(BSIZE, 1, 1);

            // Kernel to update the forces for the particles in d_i
            k_update <<< nblocks, nthreads, smem >>> (ns->d_i[g],
                                                      ns->d_p[g], // partial
                                                      ns->d_fout[g],
                                                      n_part[g], // former N
                                                      nact,
                                                      ns->e2);
        }
    }

    ns->gtime.grav_end += omp_get_wtime() - ns->gtime.grav_ini;

    ns->gtime.reduce_ini = omp_get_wtime();
    for(int g = 0; g < gpus; g++)
    {
        size_t chunk = 2<<14;
        if (n_part[g] > 0)
        {
            CSC(hipSetDevice(g));
            // Blocks, threads and shared memory configuration for the reduction.
            if (nact <= chunk) // limit 32768
            {
                dim3 rgrid   (nact,   1, 1);
                dim3 rthreads(NJBLOCK, 1, 1);

                // Kernel to reduce que temp array with the forces
                k_reduce <<< rgrid, rthreads, smem_reduce >>>(ns->d_fout[g],
                                                            ns->d_fout_tmp[g],
                                                            0,
                                                            0);
            }
            else
            {

                int smax = std::ceil(nact/(float)chunk);
                unsigned int shift = 0;
                size_t size_launch = 0;

                for(unsigned int s = 0; shift < nact; s++)
                {
                    // shift_id : s
                    // shift: moving pointer
                    // size_launch: chunk to multipy by Forces size
                    if (nact < shift + chunk)
                        size_launch = nact-shift;
                    else
                        size_launch = chunk;

                    dim3 rgrid   (size_launch,   1, 1);
                    dim3 rthreads(NJBLOCK, 1, 1);
                    k_reduce <<< rgrid, rthreads, smem_reduce >>>(ns->d_fout[g],
                                                                  ns->d_fout_tmp[g]+shift,
                                                                  s,
                                                                  shift);


                    shift += chunk;
                }
            }
        }
    }
    ns->gtime.reduce_end += omp_get_wtime() - ns->gtime.reduce_ini;

    for(int g = 0; g < gpus; g++)
    {
        if (n_part[g] > 0)
        {
            CSC(hipSetDevice(g));
            size_t chunk = nact*sizeof(Forces);

            // Copy from the GPU the new forces for the d_i particles.
            //CSC(hipMemcpy(ns->h_fout_gpu[g], ns->d_fout_tmp[g], chunk, hipMemcpyDeviceToHost));
            CSC(hipMemcpyAsync(ns->h_fout_gpu[g], ns->d_fout_tmp[g], chunk, hipMemcpyDeviceToHost, 0));
        }
    }

    // Update forces in the host
    ns->gtime.reduce_forces_ini = omp_get_wtime();
    #pragma omp parallel for
    for (int i = 0; i < nact; i++)
    {
        int id = ns->h_move[i];
        ns->h_f[id].a[0] = 0.0;
        ns->h_f[id].a[1] = 0.0;
        ns->h_f[id].a[2] = 0.0;
        ns->h_f[id].a1[0] = 0.0;
        ns->h_f[id].a1[1] = 0.0;
        ns->h_f[id].a1[2] = 0.0;

        for(int g = 0; g < gpus; g++)
        {
            if (n_part[g] > 0)
            {
                ns->h_f[id] += ns->h_fout_gpu[g][i];
            }
        }
    }
    ns->gtime.reduce_forces_end += omp_get_wtime() - ns->gtime.reduce_forces_ini;

    // Timer end
    ns->gtime.update_end += (omp_get_wtime() - ns->gtime.update_ini);
}

/** Method in charge of calculating the potential and kinetic energy
 * on the GPU devices
 */
double Hermite4GPU::get_energy_gpu()
{
    double time_energy_ini = omp_get_wtime();

    size_t d4_size = ns->n * sizeof(double4);

    for(int g = 0; g < gpus; g++)
    {
        CSC(hipSetDevice(g));

        CSC(hipMemcpyAsync(ns->d_r[g], ns->h_r, d4_size, hipMemcpyHostToDevice, 0));
        CSC(hipMemcpyAsync(ns->d_v[g], ns->h_v, d4_size, hipMemcpyHostToDevice, 0));
    }

    int nthreads = BSIZE;
    for(int g = 0; g < gpus; g++)
    {
        CSC(hipSetDevice(g));

        int nblocks = std::ceil(n_part[g]/(float)nthreads);
        k_energy <<< nblocks, nthreads >>> (ns->d_r[g],
                                            ns->d_v[g],
                                            ns->d_ekin[g],
                                            ns->d_epot[g],
                                            ns->n,
                                            n_part[g],
                                            g);
    }

    for(int g = 0; g < gpus; g++)
    {
        CSC(hipSetDevice(g));

        size_t chunk = n_part[g]*sizeof(double);
        size_t slice = g*n_part[g-1];

        CSC(hipMemcpyAsync(&ns->h_ekin[slice], ns->d_ekin[g], chunk, hipMemcpyDeviceToHost, 0));
        CSC(hipMemcpyAsync(&ns->h_epot[slice], ns->d_epot[g], chunk, hipMemcpyDeviceToHost, 0));
    }

    // Reduction on CPU
    ns->en.kinetic = 0.0;
    ns->en.potential = 0.0;
    for (int i = 0; i < ns->n; i++)
    {
        ns->en.kinetic   += ns->h_ekin[i];
        ns->en.potential += ns->h_epot[i];
    }

    double time_energy_end = omp_get_wtime() - time_energy_ini;

    return ns->en.kinetic + ns->en.potential;
}

/** Method that get the last kernel error if the code is running with the DEBUG
 * flag
 */
void Hermite4GPU::get_kernel_error()
{
    #ifdef KERNEL_ERROR_DEBUG
    logger->log_error(std::string(hipGetErrorString(hipGetLastError())));
    #endif
}

/** Method to start the device timer
 */
void Hermite4GPU::gpu_timer_start(){
    hipEventRecord(start);
}

/** Method that ends the device timer
 */
float Hermite4GPU::gpu_timer_stop(std::string f){
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float msec = 0;
    hipEventElapsedTime(&msec, start, stop);
    #if KERNEL_TIME
    if (f != "")
    {
        std::string s = "";
        s += std::string("Kernel ");
        s += std::string(SSTR(f));
        s += std::string(" : ");
        s += std::string(SSTR(msec));
        logger->log_info(s)
    }
    #endif
    return msec;
}

/** This method is not implemented becasue we use a CUDA kernel
 * to perfom the force calculation, not a host method.
 rkarim: copied the CPU version in because we're using it now
 */
void Hermite4GPU::force_calculation(const Predictor &pi, const Predictor &pj, Forces &fi) {
  double rx = pj.r[0] - pi.r[0];
  double ry = pj.r[1] - pi.r[1];
  double rz = pj.r[2] - pi.r[2];

  double vx = pj.v[0] - pi.v[0];
  double vy = pj.v[1] - pi.v[1];
  double vz = pj.v[2] - pi.v[2];

  double r2     = rx*rx + ry*ry + rz*rz + ns->e2;
  double rinv   = 1.0/sqrt(r2);
  double r2inv  = rinv  * rinv;
  double r3inv  = r2inv * rinv;
  double r5inv  = r2inv * r3inv;
  double mr3inv = r3inv * pj.m;
  double mr5inv = r5inv * pj.m;

  double rv = rx*vx + ry*vy + rz*vz;

  fi.a[0] += (rx * mr3inv);
  fi.a[1] += (ry * mr3inv);
  fi.a[2] += (rz * mr3inv);

  fi.a1[0] += (vx * mr3inv - (3 * rv ) * rx * mr5inv);
  fi.a1[1] += (vy * mr3inv - (3 * rv ) * ry * mr5inv);
  fi.a1[2] += (vz * mr3inv - (3 * rv ) * rz * mr5inv);
}

/**
BELOW HERE IS CPU ONLY, copied from cpu/Hermite4CPU.cpp
**/

/** Method that call the force_calculation method for every \f$i-\f$ and \f$j\f$
 * particles interaction of the \f$N_{act}\f$ ones.
 */
void Hermite4GPU::update_acc_jrk_cpu(unsigned int nact)
{
    ns->gtime.update_ini = omp_get_wtime();
    unsigned int i = 0;
    unsigned int j = 0;
    #pragma omp parallel for private(i,j)
    for (unsigned int k = 0; k < nact; k++)
    {
        i = ns->h_move[k];
        ns->h_f[i].a[0]  = 0.0;
        ns->h_f[i].a[1]  = 0.0;
        ns->h_f[i].a[2]  = 0.0;
        ns->h_f[i].a1[0] = 0.0;
        ns->h_f[i].a1[1] = 0.0;
        ns->h_f[i].a1[2] = 0.0;

        #pragma omp parallel for
        for (j = 0; j < ns->n; j++)
        {
            if(i == j) continue;
            force_calculation(ns->h_p[i], ns->h_p[j], ns->h_f[i]);
            // #ifdef PN
            // force_calculation_pn(ns->h_p[i], ns->h_p[j], ns->h_f[i], ns->h_f[j], i);
            // #endif
        }
    }
    ns->gtime.update_end += omp_get_wtime() - ns->gtime.update_ini;
}
